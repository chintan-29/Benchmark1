
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MyKernel() {
}
__global__ void MyKernelFlops(float n, float a, float b, float c) {
  int i =0;
  while (i<n) {
    a+=b*c;
    i++;
  }
}
__global__ void MyKernelIops(int n, int a, int b, int c) {
  int i =0;
  while (i<n) {
    a+=b*c;
    i++;
  }
  
}

void measureInIops() {
  int n = 1000000;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  MyKernelIops<<<2, 1024>>>(n, 2, 3, 6);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTime in milliseconds : %f",milliseconds);
  float giops = (n*1)/milliseconds/1e6;
  printf("\nGIOPS : %f",giops);
}

void measureInFlops() {
  int n = 1000000;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  MyKernelFlops<<<2, 1024>>>(n, 2.1f, 3.5f, 6.0f);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTime in milliseconds : %f",milliseconds);
  float gflops = (n*1)/milliseconds/1e6;
  printf("\nGFLOPS : %f",gflops);
}
int main(void)
{
  int n, type ;
  float *d_a, *d_b;
  printf("Please select from below options :\n1 -> Measure GPU Speed \n2 -> Measure memory bandwidth \n --> : ");
  scanf("%d",&type);
  if(type == 1) {
    int m;
    printf("Please select from below options :\n1 -> Measure in GIOPS \n2 -> Measure in GFLOPS \n --> : ");
    scanf("%d",&m);
    if(m==1) {
     measureInIops();
    } else if(m == 2) {
     measureInFlops();
    }
  }else if(type == 2) {
    printf("Enter Block size : ");
    scanf("%d",&n);
  
    hipMalloc(&d_a, n); 
    hipMalloc(&d_b, n);
  
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  
    hipMemcpy(d_a, d_b, n, hipMemcpyDeviceToDevice);
    
    hipEventRecord(start);
  
    MyKernel<<<2, 1024>>>();
  
    hipEventRecord(stop);
  
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
  
    printf("Bandwidth (GB/s): %fn", n*4/milliseconds/1e6);
  }
}